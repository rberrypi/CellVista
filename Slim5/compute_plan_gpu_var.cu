#include "hip/hip_runtime.h"
//#include "stdafx.h"
#include "Thrust_GPU_Var.h"
//https://github.com/thrust/thrust/blob/master/examples/summary_statistics.cu


#include <thrust/device_vector.h>
//#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
//#include <thrust/extrema.h>
//#include <cmath>
//#include <limits>

// This example computes several statistical properties of a data
// series in a single reduction.  The algorithm is described in detail here:
// http://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Parallel_algorithm
//
// Thanks to Joseph Rhoads for contributing this example


// structure used to accumulate the moments and other 
// statistical properties encountered so far.
template <typename T>
struct summary_stats_data
{
	T n;
	//T min;
	//T max;
	T mean;
	T m2;
	//T M3;
	//T M4;

	// initialize to the identity element
	void initialize()
	{
		n = mean = m2 = 0;// = M3 = M4 = 0;
		//min = std::numeric_limits<T>::max();
		//max = std::numeric_limits<T>::min();
	}

	T variance() { return m2 / (n - 1); }
	T variance_n() { return m2 / n; }
	//T skewness()   { return std::sqrt(n) * M3 / std::pow(M2, (T) 1.5); }
	//T kurtosis()   { return n * M4 / (M2 * M2); }
};

// stats_unary_op is a functor that takes in a value x and
// returns a variace_data whose mean value is initialized to x.
template <typename T>
struct summary_stats_unary_op
{
	__host__ __device__
		summary_stats_data<T> operator()(const T& x) const
	{
		summary_stats_data<T> result;
		result.n = 1;
		//result.min = x;
		//result.max = x;
		result.mean = x;
		result.m2 = 0;
		//result.M3 = 0;
		//result.M4 = 0;

		return result;
	}
};

// summary_stats_binary_op is a functor that accepts two summary_stats_data 
// structs and returns a new summary_stats_data which are an
// approximation to the summary_stats for 
// all values that have been agregated so far
template <typename T>
struct summary_stats_binary_op
	: public thrust::binary_function<const summary_stats_data<T>&,
	const summary_stats_data<T>&,
	summary_stats_data<T> >
{
	__host__ __device__
		summary_stats_data<T> operator()(const summary_stats_data<T>& x, const summary_stats_data <T>& y) const
	{
		summary_stats_data<T> result;

		// precompute some common subexpressions
		T n = x.n + y.n;
		//T n2 = n  * n;
		//T n3 = n2 * n;

		T delta = y.mean - x.mean;
		T delta2 = delta * delta;
		//T delta3 = delta2 * delta;
		//T delta4 = delta3 * delta;

		//Basic number of samples (n), min, and max
		result.n = n;
		//result.min = thrust::min(x.min, y.min);
		//result.max = thrust::max(x.max, y.max);

		result.mean = x.mean + delta * y.n / n;

		result.m2 = x.m2 + y.m2;
		result.m2 += delta2 * x.n * y.n / n;
		/*
		result.M3 = x.M3 + y.M3;
		result.M3 += delta3 * x.n * y.n * (x.n - y.n) / n2;
		result.M3 += (T) 3.0 * delta * (x.n * y.M2 - y.n * x.M2) / n;

		result.M4 = x.M4 + y.M4;
		result.M4 += delta4 * x.n * y.n * (x.n * x.n - x.n * y.n + y.n * y.n) / n3;
		result.M4 += (T) 6.0 * delta2 * (x.n * x.n * y.M2 + y.n * y.n * x.M2) / n2;
		result.M4 += (T) 4.0 * delta * (x.n * y.M3 - y.n * x.M3) / n;
		*/
		return result;
	}
};

float thrust_gpu_var::get_variance(const thrust::device_vector<float>::iterator start, const thrust::device_vector<float>::iterator stop) const
{
	typedef float T;
	summary_stats_unary_op<T>  unary_op;
	summary_stats_binary_op<T> binary_op;
	summary_stats_data<T>      init;
	init.initialize();
	auto result = thrust::transform_reduce(start, stop, unary_op, init, binary_op);
	auto val = result.variance();
	return val;
}
